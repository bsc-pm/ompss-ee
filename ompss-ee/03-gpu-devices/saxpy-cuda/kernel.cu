#include "hip/hip_runtime.h"
#include <kernel.h>

__global__ void saxpy(int n, float a, float* x, float* y)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if(i < n) y[i] = a * x[i] + y[i];
}
