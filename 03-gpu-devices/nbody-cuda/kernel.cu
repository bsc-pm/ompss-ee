#include "hip/hip_runtime.h"
#include <kernel.h>


__device__ void calculate_force(
         Particle* this_particle1, Particle* this_particle2,
         float* force_x, float* force_y, float* force_z)
				{ /* Particle_calculate_force */
					float difference_x, difference_y, difference_z;
					float distance_squared, distance;
					float force_magnitude;

					difference_x =
						this_particle2->position_x - this_particle1->position_x;
					difference_y =
						this_particle2->position_y - this_particle1->position_y;
					difference_z =
						this_particle2->position_z - this_particle1->position_z;

					distance_squared = difference_x * difference_x +
									   difference_y * difference_y +
									   difference_z * difference_z;

					distance = sqrtf(distance_squared);

					force_magnitude =
						gravitational_constant *
						(this_particle1->mass) * (this_particle2->mass) / distance_squared;

					*force_x = (force_magnitude / distance) * difference_x;
					*force_y = (force_magnitude / distance) * difference_y;
					*force_z = (force_magnitude / distance) * difference_z;
}


__global__ void calculate_force_func(int size, float time_interval,  int number_of_particles, 
                                              Particle* d_particles, Particle *output, 
											  int first_local, int last_local)
{
	int id = (blockDim.x * blockIdx.x) + threadIdx.x + first_local;
	if (id > last_local ) return;
	
	Particle* this_particle = output + id - first_local;
	
	float force_x = 0.0f, force_y = 0.0f, force_z = 0.0f;
	float total_force_x = 0.0f, total_force_y = 0.0f, total_force_z = 0.0f;
	
	for (int i = 0; i < number_of_particles; i++) {
		if (i != id) {
			calculate_force(d_particles + id, d_particles + i, &force_x, &force_y, &force_z);
			
			total_force_x += force_x;
			total_force_y += force_y;
			total_force_z += force_z;
		}
	}
	
        float velocity_change_x, velocity_change_y, velocity_change_z;
        float position_change_x, position_change_y, position_change_z;

	this_particle->mass = d_particles[id].mass;
        
        velocity_change_x =
          total_force_x * (time_interval / this_particle->mass);
        velocity_change_y =
          total_force_y * (time_interval / this_particle->mass);
        velocity_change_z =
          total_force_z * (time_interval / this_particle->mass);

	position_change_x =
	  d_particles[id].velocity_x + velocity_change_x * (0.5 * time_interval)
;
	position_change_y =
	  d_particles[id].velocity_y + velocity_change_y * (0.5 * time_interval)
;
	position_change_z =
	  d_particles[id].velocity_z + velocity_change_z * (0.5 * time_interval)
;

	this_particle->velocity_x = d_particles[id].velocity_x + velocity_change_x;
	this_particle->velocity_y = d_particles[id].velocity_y + velocity_change_y;
	this_particle->velocity_z = d_particles[id].velocity_z + velocity_change_z;

	this_particle->position_x = d_particles[id].position_x + position_change_x;
	this_particle->position_y = d_particles[id].position_y + position_change_y;
	this_particle->position_z = d_particles[id].position_z + position_change_z;

}
