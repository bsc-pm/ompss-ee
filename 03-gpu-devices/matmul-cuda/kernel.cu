#include "hip/hip_runtime.h"
#include <kernel.h>

// Thread block size
#define BLOCK_SIZE 16

// Device multiplication function called by Mul() 
// Compute C = A * B 
//	wA is the width of A 
//	wB is the width of B
__global__ void Muld(REAL* A, REAL* B, int wA, int wB, REAL* C, int NB)
{
   // Block and thread index 
   int bx = blockIdx.x;
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   // Index of the first/last sub-matrix of A processed by the block 
   int aBegin = wA * BLOCK_SIZE * by;
   int aEnd   = aBegin + wA - 1;

   // Step size used to iterate through the sub-matrices of A 
   int aStep = BLOCK_SIZE;
   // Index of the first sub-matrix of B processed by the block 
   int bBegin = BLOCK_SIZE * bx;
   // Step size used to iterate through the sub-matrices of B 
   int bStep = BLOCK_SIZE * wB;

   // The element of the block sub-matrix that is computed  by the thread 
   REAL Csub = 0;

   // Loop over all sub-matrices (A & B) required to compute the block sub-matrix 
   for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {

      // Shared memory for sub-matrix of A & B
      __shared__ REAL As[BLOCK_SIZE][BLOCK_SIZE];
      __shared__ REAL Bs[BLOCK_SIZE][BLOCK_SIZE];

      // Load the matrices from global memory to shared memory;  
      // each thread loads one element of each matrix 
      As[ty][tx] = A[a + wA * ty + tx];  
      Bs[ty][tx] = B[b + wB * ty + tx];

      // Synchronize to make sure the matrices are loaded 
      __syncthreads();

      // Multiply the two matrices together; each thread computes one element 
      // of the block sub-matrix  
      for (int k = 0; k < BLOCK_SIZE; ++k)
         Csub += As[ty][k] * Bs[k][tx];

      // Synchronize to make sure that the preceding computation is done before
      // loading two new sub-matrices of A and B in the next iteration
      __syncthreads();
   }

   // Write the block sub-matrix to global memory; each thread writes one element
   int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
   C[c + wB*ty + tx] += Csub;
}
